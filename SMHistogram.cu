
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <fstream>
#include <iostream>
#include <numeric>
#include <vector>

using std::accumulate;
using std::cout;
using std::generate;
using std::ios;
using std::ofstream;
using std::vector;

constexpr int BINS = 7;
constexpr int DIV = ((26 + BINS - 1) / BINS);

__global__ void Histogram(char *a, int *result, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ int s_result[BINS];

  if (threadIdx.x < BINS) {
    s_result[threadIdx.x] = 0;
  }

  __syncthreads();

  int alpha_position;
  for (int i = tid; i < N; i += (gridDim.x * blockDim.x)) {
    alpha_position = a[i] - 'a';
    atomicAdd(&s_result[(alpha_position / DIV)], 1);
  }

  __syncthreads();

  if (threadIdx.x < BINS) {
    atomicAdd(&result[threadIdx.x], s_result[threadIdx.x]);
  }
}


int main() {
  int N = 1 << 24;

  vector<char> h_input(N);

  vector<int> h_result(BINS);

  srand(1);
  generate(begin(h_input), end(h_input), []() { return 'a' + (rand() % 26); });

  char *d_input;
  int *d_result;
  hipMalloc(&d_input, N);
  hipMalloc(&d_result, BINS * sizeof(int));

  hipMemcpy(d_input, h_input.data(), N, hipMemcpyHostToDevice);
  hipMemcpy(d_result, h_result.data(), BINS * sizeof(int),
             hipMemcpyHostToDevice);

  int THREADS = 512;

  int BLOCKS = N / THREADS;

  Histogram<<<BLOCKS, THREADS>>>(d_input, d_result, N);

  hipMemcpy(h_result.data(), d_result, BINS * sizeof(int),
             hipMemcpyDeviceToHost);

  assert(N == accumulate(begin(h_result), end(h_result), 0));

  ofstream output_file;
  output_file.open("histogram.dat", ios::out | ios::trunc);
  for (auto i : h_result) {
    output_file << i << " \n\n";
  }
  output_file.close();

  hipFree(d_input);
  hipFree(d_result);

  return 0;
}