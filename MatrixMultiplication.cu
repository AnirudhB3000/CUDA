
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

__global__ void MatrixMultiplication(const int *a, const int *b, int *c, int N) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  c[row * N + col] = 0;

  for (int k = 0; k < N; k++) {
    c[row * N + col] += a[row * N + k] * b[k * N + col];
  }
}

void verify_result(vector <int> &a, vector <int> &b, vector <int> &c, int N) {
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      int temp = 0;
      for (int k = 0; k < N; k++) {
        temp += a[i * N + k] * b[k * N + j];
      }
      assert(temp == c[i * N + j]);
    }
  }
}

int main() {
  int N = 1 << 10;
  size_t bytes = N * N * sizeof(int);

  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);

  generate(h_a.begin(), h_a.end(), []() {return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() {return rand() % 100; });

  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

  int THREADS = 32;
  int BLOCKS = N/ THREADS;

  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  MatrixMultiplication<<<blocks, threads>>>(d_a, d_b, d_c, N);

  hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

  verify_result(h_a, h_b, h_c, N);

  cout << "Done";

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}