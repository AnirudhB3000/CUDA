
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

const int N = 1 << 10;
const int SHMEM_SIZE = 1 << 10;

__global__ void MatrixMultiplication(const int *a, const int *b, int *c) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;


  __shared__ int s_a[SHMEM_SIZE]; //static alloc
  __shared__ int s_b[SHMEM_SIZE];

  int tmp = 0;

  for (int i = 0; i < N; i += blockDim.x) {
    s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * N + i + threadIdx.x]; //load elements into tile
    s_b[threadIdx.y * blockDim.x + threadIdx.x] =
        b[i * N + threadIdx.y * N + col];


    __syncthreads(); //sync all async threads


    for (int j = 0; j < blockDim.x; j++) { // main loop to multiply
      tmp +=
          s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
    }


    __syncthreads(); // sync all async threads
  }

  c[row * N + col] = tmp;
}

void verify_result(vector <int> &a, vector <int> &b, vector <int> &c, int N) {
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      int temp = 0;
      for (int k = 0; k < N; k++) {
        temp += a[i * N + k] * b[k * N + j];
      }
      assert(temp == c[i * N + j]);
    }
  }
}

int main() {
  int N = 1 << 10;
  size_t bytes = N * N * sizeof(int);

  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);

  generate(h_a.begin(), h_a.end(), []() {return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() {return rand() % 100; });

  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

  int THREADS = 32;
  int BLOCKS = N/ THREADS;

  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  MatrixMultiplication<<<blocks, threads>>>(d_a, d_b, d_c);

  hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

  verify_result(h_a, h_b, h_c, N);

  cout << "Done";

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}