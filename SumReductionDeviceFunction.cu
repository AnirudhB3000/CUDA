//modulo operation removed, bank conflicts avoided, idle threads reduced, device function is used in last iteration to save useless work

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cassert>
#include <numeric>

using std::accumulate;
using std::generate;
using std::cout;
using std::vector;

#define SIZE 256
#define SHMEM_SIZE 256 * 4


__device__ void warpReduce(volatile int *shmem_ptr, int t) {
  shmem_ptr[t] += shmem_ptr[t + 32];
  shmem_ptr[t] += shmem_ptr[t + 16];
  shmem_ptr[t] += shmem_ptr[t + 8];
  shmem_ptr[t] += shmem_ptr[t + 4];
  shmem_ptr[t] += shmem_ptr[t + 2];
  shmem_ptr[t] += shmem_ptr[t + 1];
}
__global__ void sumReduction (int *v, int *v_r) {
    __shared__ int partial_sum[SHMEM_SIZE];

  int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

  partial_sum[threadIdx.x] = v[i] + v[i + blockDim.x];
  __syncthreads();

  for (int s = blockDim.x / 2; s > 32; s >>= 1) {
    // Each thread does work unless it is further than the stride
    if (threadIdx.x < s) {
      partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
    }
    __syncthreads();
  }

  if (threadIdx.x < 32) {
    warpReduce(partial_sum, threadIdx.x);
  }

  if (threadIdx.x == 0) {
    v_r[blockIdx.x] = partial_sum[0];
  }
}

int main() {
  int N = 1 << 16;
  size_t bytes = N * sizeof(int);

  vector<int>h_v(N);
  vector<int>h_v_r(N);

  generate(begin(h_v), end(h_v), [](){ return rand() % 10; });

  int *d_v, *d_v_r;
  hipMalloc(&d_v, bytes);
  hipMalloc(&d_v_r, bytes);

  hipMemcpy(d_v, h_v.data(), bytes, hipMemcpyHostToDevice);

  const int TB_SIZE = SIZE;

  int GRID_SIZE = N / TB_SIZE / 2;

  sumReduction<<<GRID_SIZE, TB_SIZE>>>(d_v, d_v_r);
  sumReduction<<<1, TB_SIZE>>>(d_v_r, d_v_r);

  hipMemcpy(h_v_r.data(), d_v_r, bytes, hipMemcpyDeviceToHost);

  assert(h_v_r[0] == std::accumulate(begin(h_v), end(h_v), 0));

  cout << "Done";

  return 0;
}